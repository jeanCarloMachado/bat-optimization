
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int *device;
    struct hipDeviceProp_t *deviceProp;

    device = (int *) malloc(sizeof(int));
    deviceProp = (hipDeviceProp_t *) malloc(sizeof(hipDeviceProp_t));

    hipGetDevice(device);
    hipGetDeviceProperties(deviceProp, *device);

    printf("Name: %s\n", deviceProp->name);
    printf("Global memory: %u\n", deviceProp->totalGlobalMem);
    printf("Threads per block: %u\n", deviceProp->maxThreadsPerBlock);
    printf("Shared memory per block: %u\n", deviceProp->sharedMemPerBlock);
    printf("Max grid size: %u\n", deviceProp->maxGridSize);
}
