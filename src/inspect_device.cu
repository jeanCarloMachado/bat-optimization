
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int *device;
    struct hipDeviceProp_t *deviceProp;

    device = (int *) malloc(sizeof(int));
    deviceProp = (hipDeviceProp_t *) malloc(sizeof(hipDeviceProp_t));

    hipGetDevice(device);
    hipGetDeviceProperties(deviceProp, *device);


    printf("Name: %s\n", deviceProp->name);
    printf("Global memory: %u\n", deviceProp->totalGlobalMem);
}
