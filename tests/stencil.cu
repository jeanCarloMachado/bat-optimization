#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void stencil_1d(int *in, int *out)
{
	__shared__ int temp[BLOCK_SIZE + 2  * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	
	temp[lindex] = in[gindex];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] =  in[gindex - RADIUS];
		temp[lindex + BLOCKSIZE] = in[gindex + BLOCKSIZE];
	}

	__syncthreads();

	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
		result+= temp[lindex + offset];

	out[gindex] = result;
}
