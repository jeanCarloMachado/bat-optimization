#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 512

__global__ void add(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int *x, int n)
{
	for (int i = 0; i < n; i++) {
		x[i] = random();
	}
}

int main(void) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);
	srand(time(NULL));

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a,N);
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<1,N>>>(d_a, d_b, d_c);


	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	printf ("%d\n", c[0]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
