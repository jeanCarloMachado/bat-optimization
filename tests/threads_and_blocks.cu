#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N (2048)
#define M 66600
#define THREADS_PER_BLOCK 512


__global__ void add(int *a, int *b, int *c, int n) {
		int index = threadIdx.x + blockIdx.x + blockDim.x;

		if (index < n)
		c[index] = a[index] + b[index];
}

void random_ints(int *x, int n)
{
	for (int i = 0; i < n; i++) {
		x[i] = random();
	}
}


int main(void) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);
	srand(time(NULL));

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a,N);
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


	add<<<(N + M-1) / M, M>>>(d_a,d_b,d_c,N);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf ("%d\n", c[0]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}
