
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__device__ void dobra(int num)
{
	printf ("thread x2", num*2);
}

__global__ void sumAll(int *total)
{
	__shared__ int shared;
	shared+=threadIdx.x;
	dobra(threadIdx.x);

	__syncthreads();

	*total = shared;
}



int main(void) {

	int total =0;
	int *totalPtr;

	hipMalloc((void **)&totalPtr, sizeof(int));
	hipMemcpy(totalPtr, &total, sizeof(int), hipMemcpyHostToDevice);

	sumAll<<<1,1024>>>(totalPtr);

	hipMemcpy(&total, totalPtr, sizeof(int), hipMemcpyDeviceToHost);

	printf("\nTotal: %i", total);

	hipFree(totalPtr);

	return 0;
}
