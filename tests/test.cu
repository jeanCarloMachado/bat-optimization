
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void sumAll(int *total)
{
	__shared__ int shared;
	shared+=threadIdx.x;
	printf("%i\n", threadIdx.x);

	__syncthreads();

	*total = shared;
}


int main(void) {

	int total =0;
	int *totalPtr;

	hipMalloc((void **)&totalPtr, sizeof(int));
	hipMemcpy(totalPtr, &total, sizeof(int), hipMemcpyHostToDevice);

	sumAll<<<1,1024>>>(totalPtr);

	hipMemcpy(&total, totalPtr, sizeof(int), hipMemcpyDeviceToHost);

	printf("\nTotal: %i", total);

	hipFree(totalPtr);

	return 0;
}
